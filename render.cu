#include "hip/hip_runtime.h"
#include "render.h"
#include <limits.h>
#include <stdlib.h>
//#include <hiprand_uniform.h>
#include <hiprand/hiprand_kernel.h>
//#include <hip/hip_runtime_api.h>
#include <time.h>
#define TX 64
#define TY 32
#define TZ 32

int divUp(int a, int b){return (a+b-1)/b;}

hiprandState* devStates=nullptr;
__global__ void setupSeedsKernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

void setupSeeds(int m, int n, int p)
{
    int blocks=divUp(m*n*p,TX);
    setupSeedsKernel<<<blocks,TX>>>(devStates,time(nullptr));

}

__global__ void cumulatedDensityKernel(float3 o, float3 d, aabb *cells,float* d_density, int* mutex, int ns,hiprandState* globalState)
{
    const int index = blockIdx.x*blockDim.x+threadIdx.x;

    hiprandState localState = globalState[index];
    float random1=hiprand_uniform(&localState)-0.5;
   // float random2=hiprand_uniform(NULL)-0.5;
    //float random3=hiprand_uniform(NULL)-0.5;

    if(cells[index].hit(o,d,0,FLT_MAX))
    {
    //    printf("Hit!\n");
        //mutex
        bool leave=true;
        while(leave)
        {
            if (0 == (atomicCAS(mutex,0,1)))
            {
                *d_density += cells[index].density;
                leave=false;
                atomicExch(mutex, 0);
            }
        }
    }
}



float render(int i, int j, int nx, int ny, camera& cam, aabb* cells, int m, int n, int p, int ns)
{
    //printf("render %d,%d\n",i,j);
    float density;


    float u=float(i)/float(nx);
    float v=float(j)/float(ny);

    ray r=cam.get_ray(u,v);


    float* d_density=0;
    hipMalloc(&d_density,sizeof(float));
    hipMemset(d_density,0,sizeof(float));
    int *d_mutex=0;
    hipMalloc(&d_mutex,sizeof(int));
    hipMemset(d_mutex,0,sizeof(int));
    int blocks=divUp(m*n*p,TX);
    cumulatedDensityKernel<<<blocks,TX>>>(r.origin(),r.direction(), cells,d_density,d_mutex,ns,devStates);

    hipMemcpy(&density, d_density,sizeof(float),hipMemcpyDeviceToHost);
    //fprintf(stderr,"Density at pixel %d,%d: %f\n",i,j,density);
    hipFree(d_density);
    hipFree(d_mutex);

    return density;
}
