#include "hip/hip_runtime.h"
#include "render.h"
#include <limits.h>
#include <stdlib.h>
//#include <hiprand_uniform.h>

//#include <hip/hip_runtime_api.h>
#include <time.h>
#include "cuda_check_error.h"
#include "utility.h"
#define TX 64
#define TY 32
#define TZ 32




hiprandState* devStates2D=nullptr;
__global__ void setupSeeds2DKernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

void setupSeeds2D(int tx)
{
    CudaSafeCall(hipMalloc(&devStates, tx*sizeof(hiprandState)));
    setupSeeds2DKernel<<<1,tx>>>(devStates2D,time(nullptr));
    CudaCheckError();
}

__global__ void renderAll2DKernel(float *d_pixels,int nx,int ny,float3 *d_pc,int len, camera* d_cam,float radius,int *d_mutex,int ns,hiprandState* globalState)
{
    hiprandState localState = globalState[threadIdx.x];
    const int pixel_index = blockIdx.x*blockDim.x+threadIdx.x;
    const int pc_index = blockIdx.y*blockDim.y+threadIdx.y;
    if(pixel_index>=nx*ny || pc_index>=len)
        return;
    int i,j;

    i=pixel_index%nx;
    j=pixel_index/nx;

    for(int s=0;s<ns;s++)
    {
        float u,v;
        if(ns==1){
            u=float(i)/float(nx);
            v=float(j)/float(ny);
        }
        else
        {
            u=float(i+hiprand_uniform(&localState)-0.5)/float(nx);
            v=float(j+hiprand_uniform(&localState)-0.5)/float(ny);
        }
        ray r=d_cam->get_ray(u,v);
        if(r.distance_to_pt(d_pc[pc_index])<=radius)
        {
            // printf("Hit!\n");

            bool leave=true;
            while(leave)
            {
                if (0 == (atomicCAS(&d_mutex[pixel_index],0,1)))
                {
                    d_pixels[pixel_index] = d_pixels[pixel_index]+1.0/ns;
                    leave=false;
                    atomicExch(&d_mutex[pixel_index], 0);
                }
            }
        }
    }

}

