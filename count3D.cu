#include "hip/hip_runtime.h"
#include "count3D.h"
#include "utility.h"
#include <stdio.h>
#include "cuda_check_error.h"
#define TPB 64

__global__ void count3DKernel(float3 *pc, int len, float3 lower, float3 upper, int m, int n, int p,int* counts, aabb* cells,int *mutex)
{


    const int t= blockIdx.x*blockDim.x+threadIdx.x;
    if(t>=len)
        return;
    int i= (int)(pc[t].x-lower.x)/(upper.x-lower.x)*m;
    int j= (int)(pc[t].y-lower.y)/(upper.y-lower.y)*n;
    int k= (int)(pc[t].z-lower.z)/(upper.z-lower.z)*p;


    //printf("Thread %2d: point(%f,%f,%f) is in cell(%d,%d,%d)\n", t,pc[t].x,pc[t].y,pc[t].z,i,j,k);

    int cell_index=i+j*m+k*m*n;
    if(i>=m||j>=n||k>=p)
        return;
    //mutex
    bool leave=true;
    while(leave)
    {
        if (0 == (atomicCAS(&mutex[cell_index],0,1)))
        {
            counts[cell_index]++;
            cells[cell_index].density=cells[cell_index].density+1;
            //printf("counts[%d,%d,%d]=%d\n", i,j,k, counts[cell_index]);
            //printf("cells[%d,%d,%d].density=%f\n", i,j,k, cells[cell_index].density);

            leave=false;
            atomicExch(&mutex[cell_index], 0);
        }
    }
}

//m,n,p x,y,z上分成的小正方形
void count3D(const std::vector<float3>pc, int m, int n,int p, int *counts, aabb* cells)
{
    int len = pc.size();
    aabb box=point_cloud_bounds(pc);
    box.print();
    float3* d_pc;

    CudaSafeCall(hipMalloc(&d_pc, len*sizeof(float3)));
    CudaSafeCall(hipMemcpy(d_pc, &pc[0], len*sizeof(float3),hipMemcpyHostToDevice));



    int* d_counts;

    CudaSafeCall(hipMalloc(&d_counts, m*n*p*sizeof(int)));

    int blocks=(len+TPB-1)/TPB;

    int* mutex;//all threads share on mutex.
    CudaSafeCall(hipMallocManaged((void**)&mutex, m*n*p*sizeof(int)));
    CudaSafeCall(hipMemset(mutex,0,m*n*p*sizeof(int)));

    count3DKernel<<<blocks, TPB>>>(d_pc, len, box.min(), box.max(), m,n,p,d_counts,cells, mutex);
    CudaCheckError();

    CudaSafeCall(hipMemcpy(counts, d_counts, m*n*p*sizeof(int), hipMemcpyDeviceToHost));

    CudaSafeCall(hipFree(d_pc));
    CudaSafeCall(hipFree(d_counts));
    CudaSafeCall(hipFree(mutex));
}
